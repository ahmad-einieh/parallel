// to run: nvcc oddEvenSortV3.cu -o aaa.out -gencode arch=compute_50,code=sm_50 

// ahmad einieh 
// 441106017

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void oddEvenSort(int *in, int *out, int size)
{

    __shared__ bool swappedodd;
    __shared__ bool swappedeven;

    int temp;
    bool IsOdd = true;
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    while (1)
    {
        __syncthreads();
        if (IsOdd == true)
        {
            swappedodd = false;
            __syncthreads();

            if (index % 2 == 1 && index < size - 1)
            {
                if (in[index] > in[index + 1])
                {
                    temp = in[index];
                    in[index] = in[index + 1];
                    in[index + 1] = temp;
                    swappedodd = true;
                }
            }
        }
        else
        {
            swappedeven = false;
            __syncthreads();

            if (index % 2 == 0 && index < size - 1)
            {
                if (in[index] > in[index + 1])
                {
                    temp = in[index];
                    in[index] = in[index + 1];
                    in[index + 1] = temp;
                    swappedeven = true;
                }
            }
        }

        __syncthreads();
        if (!(swappedodd || swappedeven))
            break;

        IsOdd = !IsOdd;
    }

    __syncthreads();

    if (index < size)
        out[index] = in[index];
}

int main(void)
{
    int i;
    int *array, *array_sorted;
    int *device_array, *device_array_sorted;
    int n = 8; // we can change number of element in array
    int size = sizeof(int) * n;

    hipMalloc((void **)&device_array, size);
    hipMalloc((void **)&device_array_sorted, size);

    array = (int *)malloc(size);
    array_sorted = (int *)malloc(size);

    printf("Enter the unsorted numbers: (%d numbers)\n", n);

    for (i = 0; i < n; i++)
    {
        scanf("%d", &array[i]);
    }

    hipMemcpy(device_array, array, size, hipMemcpyHostToDevice);

    oddEvenSort<<<1, n>>>(device_array, device_array_sorted, n);

    hipMemcpy(array_sorted, device_array_sorted, size, hipMemcpyDeviceToHost);

    for (i = 0; i < n; i++)
    {
        printf("%d\t", array_sorted[i]);
    }

    printf("\n");

    free(array);
    free(array_sorted);
    hipFree(device_array_sorted);
    hipFree(device_array);
}