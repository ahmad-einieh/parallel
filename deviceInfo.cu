#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int main(int argc, char const *argv[])
{
    int count;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);
    printf("Device count: %d", count);
    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("\nDevice name: %s", prop.name);
        printf("\nDevice compute capability: %d.%d", prop.major, prop.minor);
        printf("\nDevice clock rate: %d", prop.clockRate);
        printf("\nDevice memory clock rate: %d", prop.memoryClockRate);
        printf("\nDevice memory bus width: %d", prop.memoryBusWidth);
        printf("\nDevice warp size: %d", prop.warpSize);
        printf("\nDevice max threads per block: %d", prop.maxThreadsPerBlock);
        printf("\nDevice max threads dim: %d %d %d", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("\nDevice max grid size: %d %d %d", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\nDevice total global mem: %zd", prop.totalGlobalMem);
        printf("\nDevice total constant mem: %zd", prop.totalConstMem);
        printf("\nDevice shared mem per block: %zd", prop.sharedMemPerBlock);
        printf("\nDevice regs per block: %d", prop.regsPerBlock);
        printf("\nDevice multiprocessor count: %d", prop.multiProcessorCount);
        printf("\nDevice L2 cache size: %d", prop.l2CacheSize);
        printf("\nDevice max threads per multiprocessor: %d", prop.maxThreadsPerMultiProcessor);
        printf("\nDevice compute mode: %d", prop.computeMode);
        printf("\nDevice concurrent kernels: %d", prop.concurrentKernels);
        printf("\nDevice async engine count: %d", prop.asyncEngineCount);
        printf("\nDevice unified addressing: %d", prop.unifiedAddressing);
        printf("\nDevice memory pitch: %zd", prop.memPitch);
        printf("\nDevice texture alignment: %zd", prop.textureAlignment);
        printf("\nDevice device overlap: %d", prop.deviceOverlap);
        printf("\nDevice kernel exec timeout enabled: %d", prop.kernelExecTimeoutEnabled);
        printf("\nDevice can map host memory: %d", prop.canMapHostMemory);
        printf("\nDevice compute capability major: %d", prop.major);
        printf("\nDevice single to double precision perf ratio: %d", prop.singleToDoublePrecisionPerfRatio);
        
    }

    int dev;
    hipGetDevice( &dev ); 
    printf( "\nID of current CUDA device: %d\n", dev ); 
    
    return 0;
}
