#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void oddStep(int* in,int* out,int size){
    int temp;
    int idx = threadidx.x + threadidx.x * blockdim.x;
    if(idx % 2 == 1 && idx < size-1){
        if (in[idx] > in[idx+1]){
            temp = in[idx];
            in[idx] = in[idx+1];
            in[idx+1] = temp;
        }
    }
    __syncthreads();
    if(idx < size){
        out[idx] = in[idx];
    }

}

// even 
__global__ void evenStep(int* in,int* out,int size){
    int temp;
    int idx = threadidx.x + threadidx.x * blockdim.x;
    if(idx % 2 == 0 && idx < size-1){
        if (in[idx] > in[idx+1]){
            temp = in[idx];
            in[idx] = in[idx+1];
            in[idx+1] = temp;
        }
    }
    __syncthreads();
    if(idx < size){
        out[idx] = in[idx];
    }

}


