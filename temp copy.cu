// nvcc temp.cu -o temp -gencode arch=compute_50,code=sm_50 

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void count3(int *a,int* result)  { 
    int id = threadIdx.x;
    //int count = 0;
     //printf("%d\n",a[id]);
    if (a[id] == 3) {
        //count++; 
        *result = *result +  1;
 }
    
    // __syncthreads();
    // printf("%d -",count);
    //*result = *result +  count;
 }

int main() { 
    int n = 8;
    int *a;
    int *d_a;
    int size = n * sizeof(int);
    a = (int *)malloc(size);
    hipMalloc((void **)&d_a, size);
    for ( int i = 0; i < n; i++)
    {
        a[i] = i;
        //printf("%d - ", a[i]);
    }
    // printf("\n");
    
    int b = 0;
    int *d_b;
    hipMalloc((void **)&d_b, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    count3<<<1, n>>>(d_a,d_b);

    hipDeviceSynchronize();
    hipMemcpy(&b, d_b, sizeof(int), hipMemcpyDeviceToHost);

    printf("\n%d\n", b);
    return 0;
}