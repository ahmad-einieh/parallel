// to run: nvcc bitonic.cu -gencode arch=compute_50,code=sm_50 

// ahmad einieh 
// 441106017

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


__device__ __host__ int ipow(int base,int exp);

__global__ void Bitoinc(int* input_array, int* output_array,int step,int stage, int size){

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int N = ipow(2, step) / ipow(2, stage - 1);
    int shift = N / 2;
    char working = (index % N) < shift;
    char ascending = (index / ipow(2, step)) % 2 == 0;

    if (index < size && working)
    {
    if(ascending){
        if (input_array[index]>input_array[index+shift])
        {
            int temp = input_array[index];
            input_array[index]= input_array[index+shift];
            input_array[index+shift] = temp;
        }   
    }else{
        if (input_array[index]<input_array[index+shift])
        {
            int temp = input_array[index];
            input_array[index]= input_array[index+shift];
            input_array[index+shift] = temp;
        }   
    }
    }

}

int main(int argc, char** argv){

    int *array, *array_sorted;
    int *device_array , *device_array_sorted;
    int n = 16; // we can change number of element in array
    int size = sizeof(int) * n;

    array = (int*) malloc(size);
    array_sorted = (int*) malloc(size);
    
    for (int i = 0; i < n; i++)
    {
        array[i] =rand();
    }
    printf("\n");
    for (int k= 0; k < n; k++) {
        printf("%d\t",array[k]);
    }
    printf("\n");
    hipMalloc((void**) &device_array, size);
    hipMalloc((void**) &device_array_sorted, size);

    hipMemcpy(device_array,array,size,hipMemcpyHostToDevice);

    int result = ceil(log2(n));

    for (int step = 1; step <= result; step++)
    {
        for (int stage = 1; stage <= step; stage++)
        {
            Bitoinc<<<n,n>>>(device_array,device_array_sorted,step,stage,n);
        }
        
    }
    
    hipMemcpy(array_sorted, device_array, size, hipMemcpyDeviceToHost);

    for (int k= 0; k < n; k++) {
        printf("%d\t",array_sorted[k]);
    }

    printf("\n");

    free(array);
    free(array_sorted);
    hipFree(device_array_sorted);
    hipFree(device_array);

    return 0;
}

__device__ __host__ int ipow(int base, int exp)
{
    int result = 1;
    while (exp){
        if (exp % 2)
           result *= base;
        exp /= 2;
        base *= base;
    }
    return result;
}