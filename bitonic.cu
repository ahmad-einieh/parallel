// to run: nvcc bitonic.cu -o bitonic -gencode arch=compute_50,code=sm_50 

// ahmad einieh 
// 441106017

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

//power function for int
__device__ int intPower(int base, int exp)
{
    int result = 1;
    while (exp){
        if (exp % 2)
           result *= base;
        exp /= 2;
        base *= base;
    }
    return result;
}

// device function for bitonic sort
__global__ void Bitoinc(int* input_array,int step,int stage, int size){

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int N = intPower(2, step) / intPower(2, stage - 1);
    int shift = N / 2;

    //check if the index is in the range of the array
    if (index < size &&  (index % N) < shift)
    {
        // check if ascending or descending part
    if((index / intPower(2, step)) % 2 == 0){
        if (input_array[index]>input_array[index+shift])
        {
            // switch between elements
            int temp = input_array[index];
            input_array[index]= input_array[index+shift];
            input_array[index+shift] = temp;
        }   
    }else{
        if (input_array[index]<input_array[index+shift])
        {
            // switch between elements
            int temp = input_array[index];
            input_array[index]= input_array[index+shift];
            input_array[index+shift] = temp;
        }   
    }
    }

}

int main(int argc, char** argv){

    int *array, *array_sorted;
    int *device_array;
    // size of elements in array
    int n = 32; // we can change number of element in array
    int size = sizeof(int) * n;

    // allocate memory for array
    array = (int*) malloc(size);
    array_sorted = (int*) malloc(size);
    // add element to array by random value
    for (int i = 0; i < n; i++)
    {
        array[i] =rand();
    }
    // print unsorted array
    printf("\nnot sorted array:\n");
    for (int k= 0; k < n; k++) {
        printf("%d\t",array[k]);
    }
    printf("\n");

    // allocate memory for device array
    hipMalloc((void**) &device_array, size);
    // copy element from CPU array to GPU array
    hipMemcpy(device_array,array,size,hipMemcpyHostToDevice);

    // how many time will loop will run
    int result = ceil(log2(n));
    // loop for bitonic sort
    for (int step = 1; step <= result; step++)
    {
        for (int stage = 1; stage <= step; stage++)
        {
            // call bitonic funcition from device
            Bitoinc<<<n,n>>>(device_array,step,stage,n);
        }
        
    }
    // copy element from GPU array to CPU array
    hipMemcpy(array_sorted, device_array, size, hipMemcpyDeviceToHost);

    // print sorted array
    printf("\nsorted array:\n");
    for (int k= 0; k < n; k++) {
        printf("%d\t",array_sorted[k]);
    }
    printf("\n");

    // free spaces for all things
    free(array);
    free(array_sorted);
    hipFree(device_array);

    return 0;
}
