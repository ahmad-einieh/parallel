#include "hip/hip_runtime.h"
// to run: nvcc bitonic.cu -o bitonic.out -gencode arch=compute_50,code=sm_50 

// ahmad einieh 
// 441106017

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


int ipow(int base,int exp);

__global__ void Bitoinc(int* input_array, int* output_array,int step,int stage, int size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int powerStepStage = ipow(2,stage+step);
    
    

}

int main(int argc, char** argv){

    //printf("%d",ipow(2,3));
    int *array, *array_sorted;
    int *device_array , *device_array_sorted;
    int n = 8; // we can change number of element in array
    int size = sizeof(int) * n;

    array = (int*) malloc(size);
    array_sorted = (int*) malloc(size);
    
    printf("Enter the unsorted numbers: (%d numbers)\n", n);
    for (int i = 0; i < n; i++)
    {
        scanf("%d", &array[i]);
    }

    hipMalloc((void**) &device_array, size);
    hipMalloc((void**) &device_array_sorted, size);

    hipMemcpy(device_array,array,size,hipMemcpyHostToDevice);

    int result = int(log2(n));

    for (int step = 0; step < n; step++)
    {
        for (int stage = 0; stage < step; stage++)
        {
            Bitoinc(device_array,device_array_sorted,step,stage,n);
        }
        
    }
    
    hipMemcpy(array_sorted, device_array_sorted, size, hipMemcpyDeviceToHost);

    for (int k= 0; k < n; k++) {
        printf("%d\t",array_sorted[k]);
    }

    printf("\n");

    free(array);
    free(array_sorted);
    hipFree(device_array_sorted);
    hipFree(device_array);

    return 0;
}

int ipow(int base, int exp)
{
    int result = 1;
    for (;;)
    {
        if (exp & 1)
            result *= base;
        exp >>= 1;
        if (!exp)
            break;
        base *= base;
    }

    return result;
}