
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void count3(int *a,int* result)  { 
    int id = threadIdx.x;
    
    if (a[id] == 3) {
        result[id] = 1;
    }
    else{
        result[id] = 0;
    }
}

int main() { 
    int n = 8;
    int *a;
    int *d_a;
    int size = n * sizeof(int);
    a = (int *)malloc(size);
    hipMalloc((void **)&d_a, size);

    a[0] = 3;
    a[1] = 2;
    a[2] = 3;
    a[3] = 4;
    a[4] = 3;
    a[5] = 3;
    a[6] = 7;
    a[7] = 3;
    
    int *b;
    int *d_b;
    b = (int *)malloc(size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    count3<<<1, n>>>(d_a,d_b);

    hipDeviceSynchronize();
    hipMemcpy(b, d_b,size, hipMemcpyDeviceToHost);

    int count = 0;
    for (int i = 0; i < n; i++)
    {
        count = count + b[i];
    }
    

    printf("\n%d\n", count);
    return 0;
}