#include "hip/hip_runtime.h"
// to run: nvcc oddEvenSortV2.cu -o bbb.out -gencode arch=compute_50,code=sm_50 

// ahmad einieh 
// 441106017

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void oddEven(int* input_array, int* output_array, int size, int step) {

    int temp;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index % 2 == step && index < size - 1) {
        if (input_array[index] > input_array[index + 1]) {
            temp = input_array[index];
            input_array[index] = input_array[index + 1];
            input_array[index + 1] = temp;
        }
    }
    __syncthreads();
    if (index < size) {
        output_array[index] = input_array[index];
    }

}

int main()
{
    int *array, *array_sorted;
    int *device_array , *device_array_sorted;
    int n = 8; // we can change number of element in array
    int size = sizeof(int) * n;

    array = (int*) malloc(size);
    array_sorted = (int*) malloc(size);
    
    printf("Enter the unsorted numbers: (%d numbers)\n", n);
    int i;
    for (i = 0; i < n; i++)
    {
        scanf("%d", &array[i]);
    }

    hipMalloc((void**) &device_array, size);
    hipMalloc((void**) &device_array_sorted, size);

    hipMemcpy(device_array,array,size,hipMemcpyHostToDevice);

    int s;
    for (s = 1; s <= (n / 2); s++) {
        oddEven <<<1, n>>> ( device_array, device_array_sorted, n, 1);
        hipDeviceSynchronize();
        oddEven <<<1, n>>> (device_array_sorted, device_array, n, 0);
        hipDeviceSynchronize();
    }

    hipMemcpy(array_sorted, device_array, size, hipMemcpyDeviceToHost);

    int k;
    for (k= 0; k < n; k++) {
        printf("%d\t",array_sorted[k]);
    }

    printf("\n");

    free(array);
    free(array_sorted);
    hipFree(device_array_sorted);
    hipFree(device_array);

    return 0;
}
