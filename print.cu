// nvcc print.cu -gencode arch=compute_50,code=sm_50 

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print() { printf("Hello World!"); }

int main() { print<<<1,4>>>(); hipDeviceSynchronize(); }