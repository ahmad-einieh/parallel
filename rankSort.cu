// to run: nvcc rankSort.cu -o rankSort.out -gencode arch=compute_50,code=sm_50 

// ahmad einieh 
// 441106017

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <random>

__global__ void ranksort(int* input_array, int* output_array, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) { 
    int x = 0; 
    /* count number less than it */ 
    for (int j = 0; j < size; j++) {
    if (input_array[i] > input_array[j]) 
        x++; 
    if(input_array[i] == input_array[j] && j < i) 
        x++;
    }
    
    /* copy no. into correct place */ 
    output_array[x] = input_array[i]; 
    } 
} 

int main()
{
    int *array, *array_sorted;
    int *device_array , *device_array_sorted;
    int n = 8; // we can change number of element in array
    int size = sizeof(int) * n;

    array = (int*) malloc(size);
    array_sorted = (int*) malloc(size);
    
    //printf("Enter the unsorted numbers: (%d numbers)\n", n);
    int i;
    for (i = 0; i < n; i++)
    {
        array[i] =rand()%9;
        //scanf("%d", &array[i]);
    }
    
    int w;
    for (w= 0; w < n; w++) {
        printf("%d\t", array[w]);
    }
    printf("\n");


    hipMalloc((void**) &device_array, size);
    hipMalloc((void**) &device_array_sorted, size);

    hipMemcpy(device_array,array,size,hipMemcpyHostToDevice);

    ranksort<<<n,n>>>(device_array,device_array_sorted,n);

    hipMemcpy(array_sorted, device_array_sorted, size, hipMemcpyDeviceToHost);

    int k;
    for (k= 0; k < n; k++) {
        printf("%d\t",array_sorted[k]);
    }

    printf("\n");

    free(array);
    free(array_sorted);
    hipFree(device_array_sorted);
    hipFree(device_array);

    return 0;
}
